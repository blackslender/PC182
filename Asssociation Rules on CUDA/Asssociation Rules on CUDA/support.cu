#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include "support.h"
#include <bitset>

using namespace std;

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
bool testCudaForError()
{
	int size = 3;
	int a[3] = { 1,2,3 };
	int b[3] = { 1,2,3 };
	int c[3] = { 0,0,0 };
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Check output value
	if (!(c[0] == 2 && c[1] == 4 && c[2] == 6)) cudaStatus = hipErrorInvalidValue;

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus == hipSuccess;
}

void setbit(int arr[], int index, bool value) {
	int arrIndex = 0;
	while (index > sizeof(int)) {
		index -= sizeof(int);
		arrIndex++;
	}
	int q = 1 << index;

	arr[arrIndex] ^= q;
}

Dataset::Dataset(int maxRecords) {
	recordCount = new int(0);
	attrCount = new int(0);
	attributesIndex = new map<string, int>();
	syncHostToDevice();
	data = new int*[maxRecords];
	for (int i = 0; i < maxRecords; i++)
		hipMalloc(&(data[i]), SETSIZE * sizeof(int));
	hipMalloc(&_data, maxRecords * sizeof(int*));
	hipMemcpy(_data, data, maxRecords * sizeof(int*), hipMemcpyHostToDevice);
}

bool Dataset::newRecord(const int* recordRow) {
	int rowIndex = (*recordCount)++;
	hipError_t e = hipMemcpy(data[rowIndex], recordRow, SETSIZE * sizeof(int), hipMemcpyHostToDevice);
	syncHostToDevice();
	return (e == hipSuccess);
}

bool Dataset::newRecord(set<string> &recordSet) {
	int *currentRecord = new int[SETSIZE];
	for (int i = 0; i < SETSIZE; i++) currentRecord[i] = 0;
	set<string>::iterator it;
	for (it = recordSet.begin(); it != recordSet.end(); it++) {
		map<string, int>::iterator jt = attributesIndex->find(*it);
		int idx;
		if (jt == attributesIndex->end())
			idx = newAttribute(*it);
		else idx = jt->second;
		setbit(currentRecord, idx, 1);
	}
	for (int i = 0; i < SETSIZE; i++) {
		bitset<32> x(currentRecord[i]);
		cout << x;
	} cout << endl;
	bool result = newRecord(currentRecord);
	delete[] currentRecord;
	return result;
}

int* Dataset::recordSetToBit(set<string> &recordSet) {
	int *currentRecord = new int[SETSIZE];
	for (int i = 0; i < SETSIZE; i++) currentRecord[i] = 0;
	set<string>::iterator it;
	for (it = recordSet.begin(); it != recordSet.end(); it++) {
		map<string, int>::iterator jt = attributesIndex->find(*it);
		int idx;
		if (jt == attributesIndex->end())
			idx = newAttribute(*it);
		else idx = jt->second;
		setbit(currentRecord, idx, 1);
	}
	return currentRecord;
}

int Dataset::newAttribute(string attrName) {
	(*attributesIndex)[attrName] = (*attrCount)++;
	syncHostToDevice();
	return (*attrCount - 1);
}

bool Dataset::syncHostToDevice() {
	hipError_t e = hipMemcpy(recordCount, _recordCount, sizeof(int), hipMemcpyHostToDevice);
	if (e != hipSuccess) return false;
	e = hipMemcpy(attrCount, _attrCount, sizeof(int), hipMemcpyHostToDevice);
	if (e != hipSuccess) return false;
	return true;
}

int* Dataset::getRecord(int recordIndex) {
	int *currentRecord = new int[SETSIZE];
	hipError_t e = hipMemcpy(currentRecord, data[recordIndex], SETSIZE * sizeof(int), hipMemcpyDeviceToHost);
}

// Calculate support parallely
//		@_re: pointer to the record to be check
//		@_check: marking array
//		@_data: dataset
__global__ void calSupport(int* _re, char* _check, int** _data) {
	int idx = blockIdx.x;
	
}

double Dataset::supportRate(set<string> &record) {
	int* re = recordSetToBit(record);
	char* check = new char[*recordCount];

	int *_re; hipMalloc(&re, sizeof(int)*SETSIZE);
	int *_check; hipMalloc(&re, (*recordCount) * sizeof(char));
}