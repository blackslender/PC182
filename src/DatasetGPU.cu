#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include "DatasetGPU.h"
#include <bitset>
#include <fstream>
#include <sstream>
#include "support.h"
#include <hip/hip_runtime.h>
#include <>

using namespace std;

extern unsigned long long gputick;

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

// Helper function for using CUDA to add vectors in parallel.
bool testCudaForError()
{
	int size = 3;
	int a[3] = { 1,2,3 };
	int b[3] = { 1,2,3 };
	int c[3] = { 0,0,0 };
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Check output value
	if (!(c[0] == 2 && c[1] == 4 && c[2] == 6)) cudaStatus = hipErrorInvalidValue;

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus == hipSuccess;
}


DatasetGPU::DatasetGPU(int maxRecords) {
	bool success = true;
	recordCount = new int(0);
	attrCount = new int(0);
	attributesIndex = new map<string, int>();
	attributesList = new vector<string>();
	data = new int*[maxRecords];
	for (int i = 0; i < maxRecords; i++) 
		if (hipMalloc(&(data[i]), SETSIZE * sizeof(int)) != hipSuccess) success = false;
	
	if (hipMalloc(&_data, maxRecords * sizeof(int*)) != hipSuccess) success = false;
	if (hipMemcpy(_data, data, maxRecords * sizeof(int*), hipMemcpyHostToDevice) != hipSuccess) success = false;
	if (!success) cout << "Error while working with GPU (Constructor)n";
}

bool DatasetGPU::newRecord(const int* recordRow) {
	int rowIndex = (*recordCount)++;
	hipError_t e = hipMemcpy(data[rowIndex], recordRow, SETSIZE * sizeof(int), hipMemcpyHostToDevice);
	return (e == hipSuccess);
}

bool DatasetGPU::newRecord(set<string> &recordSet) {
	int *currentRecord = new int[SETSIZE];
	for (int i = 0; i < SETSIZE; i++) currentRecord[i] = 0;
	set<string>::iterator it;
	for (it = recordSet.begin(); it != recordSet.end(); it++) {
		map<string, int>::iterator jt = attributesIndex->find(*it);
		int idx;
		if (jt == attributesIndex->end())
			idx = newAttribute(*it);
		else idx = jt->second;
		setbit(currentRecord, idx, 1);
	}

	bool result = newRecord(currentRecord);
	delete[] currentRecord;
	return result;
}

int* DatasetGPU::recordSetToBit(set<string> &recordSet) {
	int *currentRecord = new int[SETSIZE];
	for (int i = 0; i < SETSIZE; i++) currentRecord[i] = 0;
	set<string>::iterator it;
	for (it = recordSet.begin(); it != recordSet.end(); it++) {
		map<string, int>::iterator jt = attributesIndex->find(*it);
		int idx;
		if (jt == attributesIndex->end())
			idx = newAttribute(*it);
		else idx = jt->second;
		setbit(currentRecord, idx, 1);
	}
	return currentRecord;
}

set<string>* DatasetGPU::bitToRecordSet(int arr[]) {
	set<string>* s = new set<string>();
	for (int i = 0; i < attributesList->size(); i++)
		if (getbit(arr, i)) s->insert(attributesList->at(i));
	return s;
}

int DatasetGPU::newAttribute(string attrName) {
	(*attributesIndex)[attrName] = (*attrCount)++;
	attributesList->push_back(attrName);
	return (*attrCount - 1);
}


int* DatasetGPU::getRecord(int recordIndex) {
	int *currentRecord = new int[SETSIZE];
	hipError_t e = hipMemcpy(currentRecord, data[recordIndex], SETSIZE * sizeof(int), hipMemcpyDeviceToHost);
	if (e != hipSuccess) cout << "Error while working with GPU (get record)...\n";
	return currentRecord;
}

// Calculate support parallely
//		@_re: pointer to the record to be check
//		@_check: marking array
//		@_data: DatasetGPU
__global__ void calSupport(int* _re, char* _check, int** _data) {
	int idx = blockIdx.x;
	int i = threadIdx.x;
	_check[idx] = 1;
	int q = _data[idx][i] & _re[i];
	if (q != _re[i]) _check[idx] = 0;

}

double DatasetGPU::supportRate(set<string> &record) {
	int* re = recordSetToBit(record);
	char* check = new char[*recordCount];
	int *_re; hipMalloc(&_re, sizeof(int)*SETSIZE);
	char *_check; hipMalloc(&_check, (*recordCount) * sizeof(char));
	hipError_t e;
	e = hipMemcpy(_re, re, SETSIZE * sizeof(int), hipMemcpyHostToDevice);
	if (e != hipSuccess)  cout << "Error while working with GPU (support rate)...\n";
	calSupport << <*recordCount, SETSIZE >> > (_re, _check, _data);
	e = hipMemcpy(check, _check, (*recordCount) * sizeof(char), hipMemcpyDeviceToHost);
	if (e != hipSuccess) cout << "Error while working with GPU (support rate)...\n";
	int suppCount = 0;
	for (int i = 0; i < *recordCount; i++)
		if (check[i] == 1) suppCount++;
	return 1.0*suppCount / (*recordCount);
}

double DatasetGPU::supportRate(int* record) {
	int* re = record;
	char* check = new char[*recordCount];
	int *_re; hipMalloc(&_re, sizeof(int)*SETSIZE);
	char *_check; hipMalloc(&_check, (*recordCount) * sizeof(char));
	hipError_t e;
	e = hipMemcpy(_re, re, SETSIZE * sizeof(int), hipMemcpyHostToDevice);
	if (e != hipSuccess)  cout << "Error while working with GPU (support rate)...\n";
	calSupport << <*recordCount, SETSIZE >> > (_re, _check, _data);
	e = hipMemcpy(check, _check, (*recordCount) * sizeof(char), hipMemcpyDeviceToHost);
	if (e != hipSuccess) cout << "Error while working with GPU (support rate)...\n";
	int suppCount = 0;
	for (int i = 0; i < *recordCount; i++)
		if (check[i] == 1) suppCount++;
	return 1.0*suppCount / (*recordCount);
}

double DatasetGPU::confidenceRate(set<string> &lhsSet, set<string> &rhsSet) {
	double s1 = supportRate(lhsSet);
	set<string> s;
	s.insert(lhsSet.begin(), lhsSet.end());
	s.insert(rhsSet.begin(), rhsSet.end());
	double s2 = supportRate(s);
	return s2 / s1;
}

double DatasetGPU::confidenceRate(int*lhsSet, int*rhsSet) {
	double s1 = supportRate(lhsSet);
	int* s = new int[SETSIZE];
	for (int i = 0; i < SETSIZE; i++) s[i] = lhsSet[i] | rhsSet[i];
	double s2 = supportRate(s);
	delete[] s;
	return s2 / s1;
}

DatasetGPU* DatasetGPU::readCSV(string filename) {
	ifstream iF;
	iF.open(filename, ios::in);

	string line;
	getline(iF, line);
	DatasetGPU *d = new DatasetGPU(DEFAULT_RECORDS_COUNT);
	set<string> attributesSet;

	// Read attribute names
	stringstream ss(line);
	string sname;
	getline(ss, sname, ','); // Skip the first "name" thing
	while (getline(ss, sname, ',')) {
		d->newAttribute(sname);
	}

	// Now read records
	while (getline(iF, line)) {
		stringstream ss(line);
		string value;
		set<string> currentRecord;
		int index = 0;
		getline(ss, sname, ','); // Skip the record name
		while (getline(ss, value, ',')) {
			if (value == "y" || value == "Y" || value == "1")
				currentRecord.insert(d->getAttributesSet()->at(index));
			index++;
		}

		// Print current record

		if (!d->newRecord(currentRecord)) cout << "Error while working with GPU (inserting record)...\n";
	}
	return d;
}